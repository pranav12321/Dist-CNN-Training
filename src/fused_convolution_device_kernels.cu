#include "hip/hip_runtime.h"
#include "fused_convolution_device.h"
#include "fused_device.h"
#include "transport.h"


#include "hip/hip_runtime.h"


extern int NUM_TILES_X;
extern int NUM_TILES_Y;
extern int DEVICE_ID_X;
extern int DEVICE_ID_Y;

extern network_config network_params_original;
extern network_config network_params_tile;
extern ftp_config ftp_params;

extern device_tile current_tile;
extern network_device current_device;
extern ftp_network ftp_cluster;

#ifdef GPU

__global__ void clear_elements_kernel(float* src, int batch, int depth,
                    int height_src, int width_src,
                    int start_x, int start_y,
                    int clear_height, int clear_width)
{
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    int m = blockIdx.y * blockDim.y + threadIdx.y;
    int total_tile_sample_size = width_src*height_src*depth;

    if (n < clear_width && m < clear_height){
        for(int sample_id = 0; sample_id < batch; sample_id++){
            for (int c = 0; c < depth; ++c)
            {
                src[(sample_id*total_tile_sample_size) + (c*height_src*width_src) + ((m+start_y)*width_src) + n+start_x] = 0.0;
            }
        }
    }
}

extern "C" void clear_edges_featuremap_device_gpu(network* net, int layer_idx, int NUM_TILES_Y, int NUM_TILES_X, int device_id_y, int device_id_x){

    int x_dim = net->layers[layer_idx].featuremap_in_w_with_boundry;
    int y_dim = net->layers[layer_idx].featuremap_in_h_with_boundry;
    int depth = net->layers[layer_idx].c;
    int total_tile_sample_size = x_dim*y_dim*depth;
    int batches = net->batch;

    if(layer_idx > 0){

        if(device_id_y == 0){
            int rows = net->layers[layer_idx].top_boundry_edges_featuremap;
            int cols = net->layers[layer_idx].featuremap_in_w_with_boundry;

            dim3 threadsPerBlock(16, 16);
            dim3 numBlocks((cols + threadsPerBlock.x -1) / threadsPerBlock.x, (rows+threadsPerBlock.y -1) / threadsPerBlock.y);
            clear_elements_kernel<<<numBlocks, threadsPerBlock>>>(net->layers[layer_idx-1].output_gpu, batches, depth, y_dim, x_dim, 0, 0, rows, cols);
        }

        if(device_id_x == 0){
            int rows = net->layers[layer_idx].featuremap_in_h_with_boundry;
            int cols = net->layers[layer_idx].left_boundry_edges_featuremap;

            dim3 threadsPerBlock(16, 16);
            dim3 numBlocks((cols + threadsPerBlock.x -1) / threadsPerBlock.x, (rows+threadsPerBlock.y -1) / threadsPerBlock.y);
            clear_elements_kernel<<<numBlocks, threadsPerBlock>>>(net->layers[layer_idx-1].output_gpu, batches, depth, y_dim, x_dim, 0, 0, rows, cols);
        }

        if(device_id_y == (NUM_TILES_Y - 1)){
            int rows = net->layers[layer_idx].bottom_boundry_edges_featuremap;
            int cols = net->layers[layer_idx].featuremap_in_w_with_boundry;
            int offset_y = net->layers[layer_idx].featuremap_in_h_without_boundry+net->layers[layer_idx].top_boundry_edges_featuremap;

            dim3 threadsPerBlock(16, 16);
            dim3 numBlocks((cols + threadsPerBlock.x -1) / threadsPerBlock.x, (rows+threadsPerBlock.y -1) / threadsPerBlock.y);
            clear_elements_kernel<<<numBlocks, threadsPerBlock>>>(net->layers[layer_idx-1].output_gpu, batches, depth, y_dim, x_dim, 0, offset_y, rows, cols);
        }

        if(device_id_y == (NUM_TILES_X - 1)){
            int rows = net->layers[layer_idx].featuremap_in_h_with_boundry;
            int cols = net->layers[layer_idx].right_boundry_edges_featuremap;
            int offset_x = net->layers[layer_idx].featuremap_in_w_without_boundry+net->layers[layer_idx].left_boundry_edges_featuremap;

            dim3 threadsPerBlock(16, 16);
            dim3 numBlocks((cols + threadsPerBlock.x -1) / threadsPerBlock.x, (rows+threadsPerBlock.y -1) / threadsPerBlock.y);
            clear_elements_kernel<<<numBlocks, threadsPerBlock>>>(net->layers[layer_idx-1].output_gpu, batches, depth, y_dim, x_dim, offset_x, 0, rows, cols);
        }

    }
}


extern "C" void clear_edges_delta_device_gpu(network* net, int layer_idx, int NUM_TILES_Y, int NUM_TILES_X, int device_id_y, int device_id_x){

    int x_dim = net->layers[layer_idx].delta_in_w_with_boundry;
    int y_dim = net->layers[layer_idx].delta_in_h_with_boundry;
    int depth = (net->layers[layer_idx].type == CONVOLUTIONAL) ? net->layers[layer_idx].n : net->layers[layer_idx].c;
    int total_tile_sample_size = x_dim*y_dim*depth;
    int batches = net->batch;


    if(layer_idx > 0){

        if(device_id_y == 0){
            int rows = net->layers[layer_idx].top_boundry_edges_delta;
            int cols = net->layers[layer_idx].delta_in_w_with_boundry;

            dim3 threadsPerBlock(16, 16);
            dim3 numBlocks((cols + threadsPerBlock.x -1) / threadsPerBlock.x, (rows+threadsPerBlock.y -1) / threadsPerBlock.y);
            clear_elements_kernel<<<numBlocks, threadsPerBlock>>>(net->layers[layer_idx].delta_gpu, batches, depth, y_dim, x_dim, 0, 0, rows, cols);
        }

        if(device_id_x == 0){
            int rows = net->layers[layer_idx].delta_in_h_with_boundry;
            int cols = net->layers[layer_idx].left_boundry_edges_delta;

            dim3 threadsPerBlock(16, 16);
            dim3 numBlocks((cols + threadsPerBlock.x -1) / threadsPerBlock.x, (rows+threadsPerBlock.y -1) / threadsPerBlock.y);
            clear_elements_kernel<<<numBlocks, threadsPerBlock>>>(net->layers[layer_idx].delta_gpu, batches, depth, y_dim, x_dim, 0, 0, rows, cols);
        }

        if(device_id_y == (NUM_TILES_Y - 1)){
            int rows = net->layers[layer_idx].bottom_boundry_edges_delta;
            int cols = net->layers[layer_idx].delta_in_w_with_boundry;
            int offset_y = net->layers[layer_idx].delta_in_h_without_boundry+net->layers[layer_idx].top_boundry_edges_delta;

            dim3 threadsPerBlock(16, 16);
            dim3 numBlocks((cols + threadsPerBlock.x -1) / threadsPerBlock.x, (rows+threadsPerBlock.y -1) / threadsPerBlock.y);
            clear_elements_kernel<<<numBlocks, threadsPerBlock>>>(net->layers[layer_idx].delta_gpu, batches, depth, y_dim, x_dim, 0, offset_y, rows, cols);
        }

        if(device_id_y == (NUM_TILES_X - 1)){
            int rows = net->layers[layer_idx].delta_in_h_with_boundry;
            int cols = net->layers[layer_idx].right_boundry_edges_delta;
            int offset_x = net->layers[layer_idx].delta_in_w_without_boundry+net->layers[layer_idx].left_boundry_edges_delta;

            dim3 threadsPerBlock(16, 16);
            dim3 numBlocks((cols + threadsPerBlock.x -1) / threadsPerBlock.x, (rows+threadsPerBlock.y -1) / threadsPerBlock.y);
            clear_elements_kernel<<<numBlocks, threadsPerBlock>>>(net->layers[layer_idx].delta_gpu, batches, depth, y_dim, x_dim, offset_x, 0, rows, cols);
        }

    }
}

extern "C" void clear_spurious_edges_featuremap_gpu(network* net, int layer_idx){

    int x_dim = net->layers[layer_idx].featuremap_in_w_with_boundry;
    int y_dim = net->layers[layer_idx].featuremap_in_h_with_boundry;
    int depth = net->layers[layer_idx].c;
    int total_tile_sample_size = x_dim*y_dim*depth;
    int batches = net->batch;

    int device_id_y = ftp_params.DEVICE_ID_Y;
    int device_id_x = ftp_params.DEVICE_ID_X;
    int NUM_TILES_X = ftp_params.NUM_TILES_X;
    int NUM_TILES_Y = ftp_params.NUM_TILES_Y;

    int start_x_coordinate = network_params_tile.spurious_blocks[layer_idx].start_x_coordinate;
    int start_y_coordinate = network_params_tile.spurious_blocks[layer_idx].start_y_coordinate;

    float* featuremap;

    // if(layer_idx > 0)
    //     featuremap = net->layers[layer_idx-1].output;
    // else
    featuremap = net->input;

    if((device_id_x == (NUM_TILES_X - 1)) && (start_x_coordinate > -1)){
        int rows = net->layers[layer_idx].featuremap_in_h_with_boundry;
        int cols = net->layers[layer_idx].featuremap_in_w_with_boundry - (net->layers[layer_idx].left_boundry_edges_featuremap + start_x_coordinate);
        int offset_x = net->layers[layer_idx].left_boundry_edges_featuremap + start_x_coordinate;

        dim3 threadsPerBlock(16, 16);
        dim3 numBlocks((cols + threadsPerBlock.x -1) / threadsPerBlock.x, (rows+threadsPerBlock.y -1) / threadsPerBlock.y);
        clear_elements_kernel<<<numBlocks, threadsPerBlock>>>(net->layers[layer_idx].delta_gpu, batches, depth, y_dim, x_dim, offset_x, 0, rows, cols);
    }

    if((device_id_y == (NUM_TILES_Y - 1)) && (start_y_coordinate > -1)){
        int rows = net->layers[layer_idx].featuremap_in_h_with_boundry - (net->layers[layer_idx].top_boundry_edges_featuremap + start_y_coordinate);
        int cols = net->layers[layer_idx].featuremap_in_w_with_boundry;
        int offset_y = net->layers[layer_idx].top_boundry_edges_featuremap + start_y_coordinate;

        dim3 threadsPerBlock(16, 16);
        dim3 numBlocks((cols + threadsPerBlock.x -1) / threadsPerBlock.x, (rows+threadsPerBlock.y -1) / threadsPerBlock.y);
        clear_elements_kernel<<<numBlocks, threadsPerBlock>>>(net->layers[layer_idx].delta_gpu, batches, depth, y_dim, x_dim, 0, offset_y, rows, cols);
    }
   
}

extern "C" void clear_spurious_edges_delta_gpu(network* net, int layer_idx){

    int x_dim = net->layers[layer_idx].delta_in_w_with_boundry;
    int y_dim = net->layers[layer_idx].delta_in_h_with_boundry;
    int depth = (net->layers[layer_idx].type == CONVOLUTIONAL) ? net->layers[layer_idx].n : net->layers[layer_idx].c;
    int total_tile_sample_size = x_dim*y_dim*depth;
    int batches = net->batch;

    int device_id_y = ftp_params.DEVICE_ID_Y;
    int device_id_x = ftp_params.DEVICE_ID_X;
    int NUM_TILES_X = ftp_params.NUM_TILES_X;
    int NUM_TILES_Y = ftp_params.NUM_TILES_Y;

    int start_x_coordinate = network_params_tile.spurious_blocks[layer_idx+1].start_x_coordinate;
    int start_y_coordinate = network_params_tile.spurious_blocks[layer_idx+1].start_y_coordinate;

    if((device_id_x == (NUM_TILES_X - 1)) && (start_x_coordinate > -1)){
        int rows = net->layers[layer_idx].delta_in_h_with_boundry;
        int cols = net->layers[layer_idx].delta_in_w_with_boundry - (net->layers[layer_idx].left_boundry_edges_delta + start_x_coordinate);
        int offset_x = net->layers[layer_idx].left_boundry_edges_delta + start_x_coordinate;

        dim3 threadsPerBlock(16, 16);
        dim3 numBlocks((cols + threadsPerBlock.x -1) / threadsPerBlock.x, (rows+threadsPerBlock.y -1) / threadsPerBlock.y);
        clear_elements_kernel<<<numBlocks, threadsPerBlock>>>(net->layers[layer_idx].delta_gpu, batches, depth, y_dim, x_dim, offset_x, 0, rows, cols);
    }

    if((device_id_y == (NUM_TILES_Y - 1)) && (start_y_coordinate > -1)){
        int rows = net->layers[layer_idx].delta_in_h_with_boundry - (net->layers[layer_idx].top_boundry_edges_delta + start_y_coordinate);
        int cols = net->layers[layer_idx].delta_in_w_with_boundry;
        int offset_y = net->layers[layer_idx].top_boundry_edges_delta + start_y_coordinate;

        dim3 threadsPerBlock(16, 16);
        dim3 numBlocks((cols + threadsPerBlock.x -1) / threadsPerBlock.x, (rows+threadsPerBlock.y -1) / threadsPerBlock.y);
        clear_elements_kernel<<<numBlocks, threadsPerBlock>>>(net->layers[layer_idx].delta_gpu, batches, depth, y_dim, x_dim, 0, offset_y, rows, cols);
    }
   
}

    __global__ void copy_slice_kernel(float* dst, float* src, int batch, int depth,
                    int height_src, int width_src, int height_dst, int width_dst,
                    int src_start_x, int src_start_y, int dst_start_x, int dst_start_y,
                    int copy_height_src, int copy_width_src, int copy_height_dst, int copy_width_dst,
                    float* workspace)
    {
        int w = blockIdx.x * blockDim.x + threadIdx.x;
        int h = blockIdx.y * blockDim.y + threadIdx.y;
        int total_tile_sample_size = depth*copy_height_src*copy_width_src;

        if (h < copy_height_src && w < copy_width_src){

            float* src_intermediate = src;

            if(dst == src){
                src_intermediate = workspace;

                for(int b = 0; b < batch; b++){
                    for(int d = 0; d < depth; d++){
                        workspace[b*total_tile_sample_size + d*copy_height_src*copy_width_src + h*copy_width_src + w] = 
                        src[b*depth*height_src*width_src + d*height_src*width_src + (h + src_start_y)*width_src + w + src_start_x];
                    }
                }
            }

            for(int b = 0; b < batch; b++){
                for(int d = 0; d < depth; d++){
                    dst[b*total_tile_sample_size + d*height_dst*width_dst + (h + dst_start_y)*width_dst + w + dst_start_x] = 
                    src_intermediate[b*total_tile_sample_size + d*copy_height_dst*copy_width_dst + h*copy_width_dst + w];     
                }        
            }
        }
    }

    extern "C" void copy_slice_gpu(float* dst, float* src, int batch, int depth,
                    int height_src, int width_src, int height_dst, int width_dst,
                    int src_start_x, int src_start_y, int dst_start_x, int dst_start_y,
                    int copy_height_src, int copy_width_src, int copy_height_dst, int copy_width_dst,
                    float* workspace){

        int rows = copy_height_src;
        int cols = copy_width_src;

        dim3 threadsPerBlock(16, 16);
        dim3 numBlocks((cols + threadsPerBlock.x -1) / threadsPerBlock.x, (rows+threadsPerBlock.y -1) / threadsPerBlock.y);
        copy_slice_kernel<<<numBlocks, threadsPerBlock>>>(dst, src, batch, depth,
                    height_src, width_src, height_dst, width_dst,
                    src_start_x, src_start_y, dst_start_x, dst_start_y,
                    copy_height_src, copy_width_src, copy_height_dst, copy_width_dst,
                    workspace);
    }



#endif
